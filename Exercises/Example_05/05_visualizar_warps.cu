#include <iostream>
#include <hip/hip_runtime.h>

__global__ void kernel_visualizar() {
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	printf("Hola desde el hilo %d (threadIdx.x=%d, blockIdx.x=%d)\n", tid, threadIdx.x, blockIdx.x);
}

int main(){
	const int N = 64; // Numero total de hilos

	int threads_per_block = 32; // Probaremos cambiar este valor
	int blocks = (N + threads_per_block -1) / threads_per_block;

	kernel_visualizar <<< blocks, threads_per_block >>> ();
	hipDeviceSynchronize(); // Esperar que terminen todos los hilos

	return 0;
}

// nvcc 05_visualizar_warps.cu -o visualizar_warps