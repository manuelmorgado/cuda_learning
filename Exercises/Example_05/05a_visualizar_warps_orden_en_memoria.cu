#include <iostream>
#include <hip/hip_runtime.h>

__global__ void kernel_guardar_orden(int* salida){
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    salida[tid] = tid; // Cada hilo escribe su propio ID
}

int main() {

    const int N = 64; // Numero total de hilos

    int threads_per_block = 32;
    int blocks = (N + threads_per_block - 1) / threads_per_block;

    // Reservar memoria en CPU
    int* h_salida = new int[N];

    // Reservar memoria en GPU
    int* d_salida;
    hipMalloc(&d_salida, N * sizeof(int));

    // Lanzar el kernel
    kernel_guardar_orden <<< blocks, threads_per_block >>> (d_salida);

    //Esperar a que termine el kernel
    hipDeviceSynchronize();

    // Copiar resultados de GPU al CPU
    hipMemcpy(h_salida, d_salida, N * sizeof(int), hipMemcpyDeviceToHost);

    // Imprimir resultados
    for (int i = 0; i < N; ++i){
        std::cout << h_salida[i] << " ";
    }
    std::cout<< std::endl;

    // Liberar memoria
    delete[] h_salida;
    hipFree(d_salida);

    return 0;
}

// nvcc guardar_orden.cu -o guardar_orden
