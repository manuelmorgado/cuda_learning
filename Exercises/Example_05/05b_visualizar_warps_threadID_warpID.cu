#include <iostream>
#include <hip/hip_runtime.h>

__global__ void kernel_guardar_warp_info(int* salida_tid, int* salida_warp) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;  // ID global del hilo
    int warp_id = tid / 32;                           // Warp ID: cada 32 hilos forman un warp

    salida_tid[tid] = tid;       // Guardar el Thread ID
    salida_warp[tid] = warp_id;  // Guardar el Warp ID
}

int main() {
    const int N = 64;  // Número total de hilos que queremos lanzar

    int threads_per_block = 32;  // Cambia este valor si quieres experimentar
    int blocks = (N + threads_per_block - 1) / threads_per_block;

    int* h_tid = new int[N];
    int* h_warp = new int[N];

    int *d_tid, *d_warp;
    hipMalloc(&d_tid, N * sizeof(int));
    hipMalloc(&d_warp, N * sizeof(int));

    kernel_guardar_warp_info<<<blocks, threads_per_block>>>(d_tid, d_warp);

    hipDeviceSynchronize();

    hipMemcpy(h_tid, d_tid, N * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(h_warp, d_warp, N * sizeof(int), hipMemcpyDeviceToHost);

    std::cout << "Thread ID - Warp ID:\n";
    for (int i = 0; i < N; ++i) {
        std::cout << "Hilo " << h_tid[i] << " pertenece al Warp " << h_warp[i] << "\n";
    }

    delete[] h_tid;
    delete[] h_warp;
    hipFree(d_tid);
    hipFree(d_warp);

    return 0;
}

// nvcc 05b_visualizar_warps_threadID_warpID.cu -o visualizar_warps_ordenado