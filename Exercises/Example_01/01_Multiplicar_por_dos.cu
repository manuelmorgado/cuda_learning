#include <iostream>
#include <vector>
#include <hip/hip_runtime.h>

// Kernel CUDA
__global__ void multiplicar_por_dos(int *datos, int N){
    int idx = threadIdx.x;
    if (idx < N) {
        datos[idx] *= 2;
    }
}

// Funcion principal
int main() {

    // Mide memoria libre
    size_t free_mem, total_mem;
    hipMemGetInfo(&free_mem, &total_mem);
    std::cout << "Memoria libre: " << free_mem / 1024 / 1024 << " MB\n";

    const int N = 1000000;
    std::vector<int> h_datos(N);

    // Inicializa los elementos del vector
    for (int i = 0; i<N; ++i)
        h_datos[i] = i;

    // Reservar el bloque de memoria en la GPU
    int *d_datos;
    hipMalloc(&d_datos, N * sizeof(int));

    // Mueve memory al GPU (device)
    hipMemcpy(d_datos, h_datos.data(), N * sizeof(int), hipMemcpyHostToDevice);

    // Lanza kernel
    multiplicar_por_dos<<< 1, N >>>(d_datos, N);

    // Mueve de vuelta la memoria al CPU (host)
    hipMemcpy(h_datos.data(), d_datos, N * sizeof(int), hipMemcpyDeviceToHost);

    // Muestra los resultados
    for (auto val : h_datos){
        std::cout << val << " ";
    }

    std::cout << std::endl;

    // Mide memoria libre
    hipMemGetInfo(&free_mem, &total_mem);
    std::cout << "Memoria libre: " << free_mem / 1024 / 1024 << " MB\n";

    // Liberar memoria
    hipFree(d_datos);

    // Mide memoria libre
    hipMemGetInfo(&free_mem, &total_mem);
    std::cout << "Memoria libre: " << free_mem / 1024 / 1024 << " MB\n";

    return 0;


}

//nvcc 01_Multiplicar_por_dos.cu -o por2