#include <iostream>
#include <vector>
#include <hip/hip_runtime.h>

#define CHECK_CUDA(call) \
    if ((call) != hipSuccess) { \
        std::cerr << "Error en llamada CUDA Driver API en " << __FILE__ << ":" << __LINE__ << std::endl; \
        return -1; \
    }

int main() {

	hipDevice_t cuDevice;
	hipCtx_t cuContext;
	hipModule_t cuModule;
	hipFunction_t cuFunction;

	const int N = 10;
	std::vector<int> h_datos(N);

	for (int i = 0; i < N; ++i){
		h_datos[i] = i;
	}

	// Inicializa el Driver API
	CHECK_CUDA(hipInit(0));
	CHECK_CUDA(hipDeviceGet(&cuDevice, 0));
	CHECK_CUDA(hipCtxCreate(&cuContext, 0, cuDevice));

	// Cargar module (el PTX compilado)
	CHECK_CUDA(hipModuleLoad(&cuModule, "multiplicar_por_dos_kernel.ptx"));
	CHECK_CUDA(hipModuleGetFunction(&cuFunction, cuModule, "multiplicar_por_dos"));

	// Reservar memoria en GPU
	hipDeviceptr_t d_datos;
	CHECK_CUDA(hipMalloc(&d_datos, N * sizeof(int)));

	// Copiar datos del host al device
	CHECK_CUDA(hipMemcpyHtoD(d_datos, h_datos.data(), N * sizeof(int)));

	// Perparar parametros para el kernel
	void* args[] = { (void*)&d_datos, (void*)&N };
	
	int threads_per_block = 256;
	int blocks = (N + threads_per_block - 1)/ threads_per_block;

	// Lanzar el kernel
	CHECK_CUDA(hipModuleLaunchKernel(cuFunction,
							  blocks, 1, 1,				// gridDim
							  threads_per_block, 1, 1,	// blockDim
							  0,						// shared memory
							  0,						// stream
							  args,						// argumentos
							  nullptr                   // extra
							  ));

	// Esperar a que termine el kernel
	CHECK_CUDA(hipCtxSynchronize());

	// Copiar de vuelta del device al host
	CHECK_CUDA(hipMemcpyDtoH(h_datos.data(), d_datos, N * sizeof(int)));

	// Imprimir resultados
	std::cout << "Resultados:\n";
	for  (auto val : h_datos){
		std::cout << val << " ";
	}
	std::cout << std::endl;

	// Liberar memoria
	hipFree(d_datos);
	hipModuleUnload(cuModule);
	hipCtxDestroy(cuContext);

	return 0;
}

// nvcc 04_Multiplicar_por_dos_driver_api.cu -lcuda -o driver_api