
#include <hip/hip_runtime.h>
extern "C" __global__ void multiplicar_por_dos(int *datos, int N){
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	if (idx < N){
		datos[idx] *= 2;
	}
}

// nvcc -ptx multiplicar_por_dos_kernel.cu -o multiplicar_por_dos_kernel.ptx