#include <iostream>
#include <hip/hip_runtime.h>

int main() {
    int device_count = 0;
    hipGetDeviceCount(&device_count);
    std::cout << "Número de GPUs disponibles: " << device_count << std::endl;

    for (int device = 0; device < device_count; ++device) {
        hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop, device);

        std::cout << "\n=== GPU " << device << " ===" << std::endl;
        std::cout << "Nombre: " << prop.name << std::endl;
        std::cout << "Memoria Global total: " << prop.totalGlobalMem / (1024 * 1024) << " MB" << std::endl;
        std::cout << "Multiprocesadores (SMs): " << prop.multiProcessorCount << std::endl;
        std::cout << "Máximo número de hilos por bloque: " << prop.maxThreadsPerBlock << std::endl;
        std::cout << "Tamaño máximo de un bloque de hilos: ("
                  << prop.maxThreadsDim[0] << ", "
                  << prop.maxThreadsDim[1] << ", "
                  << prop.maxThreadsDim[2] << ")" << std::endl;
        std::cout << "Tamaño máximo de una grilla: ("
                  << prop.maxGridSize[0] << ", "
                  << prop.maxGridSize[1] << ", "
                  << prop.maxGridSize[2] << ")" << std::endl;
        std::cout << "Memoria compartida por bloque: " << prop.sharedMemPerBlock / 1024 << " KB" << std::endl;
        std::cout << "Máxima cantidad de registros por bloque: " << prop.regsPerBlock << std::endl;
        std::cout << "Compute Capability: " << prop.major << "." << prop.minor << std::endl;
        std::cout << "Tamaño de warp: " << prop.warpSize << std::endl;
        std::cout << "Reloj de GPU: " << prop.clockRate / 1000 << " MHz" << std::endl;
    }

    return 0;
}

//nvcc 03_GPU_data.cu -o my_gpu