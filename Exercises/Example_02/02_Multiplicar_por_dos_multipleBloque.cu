#include <iostream>
#include <hip/hip_runtime.h>

__global__ void multiplicar_por_dos(int *datos, int N) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < N) {
        datos[idx] *= 2;
    }
}

int main() {

    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);
    std::cout << "Máximo hilos por bloque: " << prop.maxThreadsPerBlock << std::endl;

    int N = 1000;
    int *h_data = new int[N];
    int *d_data;

    // Inicializar
    for (int i = 0; i < N; ++i)
        h_data[i] = i;

    hipMalloc(&d_data, N * sizeof(int));
    hipMemcpy(d_data, h_data, N * sizeof(int), hipMemcpyHostToDevice);

    // Lanzar muchos bloques e hilos
    int threads_per_block = prop.maxThreadsPerBlock;
    int blocks = (N + threads_per_block - 1) / threads_per_block;
    multiplicar_por_dos<<<blocks, threads_per_block>>>(d_data, N);

    hipMemcpy(h_data, d_data, N * sizeof(int), hipMemcpyDeviceToHost);

    std::cout << "Primeros N resultados:\n";
    for (int i = 0; i < N; ++i)
        std::cout << h_data[i] << " ";
    std::cout << std::endl;

    hipFree(d_data);
    delete[] h_data;
    return 0;
}

//nvcc 02_Multiplicar_por_dos_multipleBloque.cu -o por2_mb